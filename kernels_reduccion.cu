
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string.h>
#include <sys/time.h>
#include <stdlib.h>     /* srand, rand */
#include <time.h>       /* time */

using namespace std;

//Poner esto a 1 para imprimir los resultados


double cpuSecond(){

	struct timeval tp;
	gettimeofday(&tp, NULL);
	return((double)tp.tv_sec + (double)tp.tv_usec*1e-6);
}
int maximo_local(int* vector,int N){

    int maximo = 0;

    for(int i = 0; i < N; i++){

        if(vector[i] > maximo)
            maximo = vector[i];

    }

    return maximo;
}


__global__ void kernel_reduccion_intervalos(int* device_vector,int* device_salida){

  extern __shared__ int datos[];

  int tid = threadIdx.x; //numero de hebra

  int posicion = blockIdx.x * blockDim.x + threadIdx.x;
  int index = 0;

  datos[tid] = device_vector[posicion];

  for(int i = 1; i > blockDim.x; i *= 2){
    index = 2 * i * tid;

    if(index < blockDim.x){

      if(datos[tid] < datos[tid+i]){

        datos[tid] = datos[tid+i];

      }

    }
      __syncthreads();
  }


  //Guardo los resultados en el vector D
    if(device_salida[blockIdx.x] == 0){

      device_salida[blockIdx.x] = datos[0];
    }

}

__global__ void kernel_reduccion_secuencial(int* device_vector,int* device_salida){

    extern __shared__ int datos[];

    int tid = threadIdx.x; //numero de hebra

    int posicion = blockIdx.x * blockDim.x + threadIdx.x;

    datos[tid] = device_vector[posicion];


    __syncthreads();

    for(int i = blockDim.x/2; i > 0; i >>= 1){

      if(tid < i){

        if(datos[tid] < datos[tid+1]){

          datos[tid] = datos[tid+1];

        }

      }
        __syncthreads();
    }


    //Guardo los resultados en el vector D
      if(device_salida[blockIdx.x] == 0){

        device_salida[blockIdx.x] = datos[0];
      }
}

__device__ void desenrrollado_reduce_32(volatile int* datos, int tid){

    if(datos[tid] < datos[tid+32]) datos[tid] = datos[tid+32];
    if(datos[tid] < datos[tid+16]) datos[tid] = datos[tid+16];
    if(datos[tid] < datos[tid+8]) datos[tid] = datos[tid+8];
    if(datos[tid] < datos[tid+4]) datos[tid] = datos[tid+4];
    if(datos[tid] < datos[tid+2]) datos[tid] = datos[tid+2];
    if(datos[tid] < datos[tid+1]) datos[tid] = datos[tid+1];


}

__global__ void kernel_reduccion_desenrrollado_parcial(int* device_vector,int* device_salida){

    extern __shared__ int datos[];

    int tid = threadIdx.x; //numero de hebra

    int posicion = blockIdx.x * blockDim.x + threadIdx.x;

    datos[tid] = device_vector[posicion];


    for(int i = blockDim.x/2; i > 32; i >>= 1){

      if(tid < i){

        if(datos[tid] < datos[tid+1]){

          datos[tid] = datos[tid+1];

        }

      }
        __syncthreads();
    }

    if(tid < 32) desenrrollado_reduce_32(datos,tid);

    //Guardo los resultados en el vector D
      if(device_salida[blockIdx.x] == 0){

        device_salida[blockIdx.x] = datos[0];
      }

}

//para bloques de 2048
__global__ void kernel_reduccion_desenrrollado_total(int* device_vector,int* device_salida){

    extern __shared__ int datos[];

    int tid = threadIdx.x; //numero de hebra

    int posicion = blockIdx.x * blockDim.x + threadIdx.x;

    datos[tid] = device_vector[posicion];

    if(blockDim.x >= 2048){
        if(tid < 1024){
            if(datos[tid] < datos[tid + 1024]){
                datos[tid] = datos[tid + 1024];
            }
        }
    }

    if(blockDim.x >= 1024){
        if(tid < 512){
            if(datos[tid] < datos[tid + 512]){
                datos[tid] = datos[tid + 512];
            }
        }
        __syncthreads();
    }

    if(blockDim.x >= 512){
        if(tid < 256){
            if(datos[tid] < datos[tid + 256]){
                datos[tid] = datos[tid + 256];
            }
        }
        __syncthreads();
    }

    if(blockDim.x >= 256){
        if(tid < 128){
            if(datos[tid] < datos[tid + 128]){
                datos[tid] = datos[tid + 128];
            }
        }
        __syncthreads();
    }
    if(blockDim.x >= 128){
        if(tid < 64){
            if(datos[tid] < datos[tid + 64]){
                datos[tid] = datos[tid + 64];
            }
        }
        __syncthreads();
    }

    if(tid < 32) desenrrollado_reduce_32(datos,tid);

    __syncthreads();
    //Guardo los resultados en el vector D
      if(device_salida[blockIdx.x] == 0){

        device_salida[blockIdx.x] = datos[0];
      }
}



int main(int argc, char* argv[]){

    bool imprimir = false;


    if(argc < 4){
        cout << "Sintaxis: ./program <Numero de kernel a ejecutar> <Tamaño del problema> <Numero de bloques> "  << endl;
        exit(-1);
    }

    //Depuracion de errores CUDA
    int devID;
    hipError_t error_cuda;

    error_cuda = hipGetDevice(&devID);
    if(error_cuda != hipSuccess){

      cout << "Error. No hay tarjeta grafica nvdia o no esta instalado el driver" << endl;
      exit(-1);

    }

    hipDeviceProp_t propiedades;
    hipGetDeviceProperties(&propiedades, devID);

    if(imprimir)
        cout << "Device " << devID << " " << propiedades.name << " con capacidad computacional: " << propiedades.major << "." << propiedades.minor << endl;

    //Kernel a ejecutar
    int kernel = atoi(argv[1]);

    //Tamaño del problema
    int N      = atoi(argv[2]);

    int bloques_por_grid  = atoi(argv[3]);

    //Memoria que es necesaria reservar para el vector device
    int device_memory = N*sizeof(int);

    int* vector = new int[N];
    int* resultado = new int[N];

    int* device_vector;
    int* device_salida;

    //Reservor la memoria para el vector device
    error_cuda = hipMalloc( (void**) &device_vector, device_memory);

    if(error_cuda != hipSuccess){

        cout << "No se ha podido reservar memoria para el vector <device_vector>" << endl;
        exit(-1);

    }

    error_cuda = hipMalloc( (void**) &device_salida, bloques_por_grid*sizeof(int));
    if(error_cuda != hipSuccess){

        cout << "No se ha podido reservar memoria para el vector <device_salida>" << endl;
        exit(-1);

    }

    //rellena el vecto con numero de 1 a N aleatorios
    int numero_random = 0;
    int maximo = -1;
    srand(time(NULL));
    for(int i = 0; i < N; i++){

        numero_random = rand() % N + 1;
        if(numero_random > maximo)
            maximo = numero_random;

        vector[i] = numero_random;

        resultado[i] = 0;
    }

    //Copio el contenido del vector a el vector device
    error_cuda = hipMemcpy(device_vector,vector,device_memory, hipMemcpyHostToDevice);
    if(error_cuda != hipSuccess){

        cout << "No se pudo copair el contenido de <vector> a el vector device <device_vector>" << endl;
        exit(-1);

    }

    //Pongo a 0 todas las casillas del vector <<vector_salida>>
    error_cuda = hipMemcpy(device_salida,resultado, bloques_por_grid*sizeof(int), hipMemcpyHostToDevice);
    if(error_cuda != hipSuccess){

        cout << "No se pudo copair el contenido de <resultado> a el vector device <device_salida>" << endl;
        exit(-1);

    }

    int hebras = ceil(N/bloques_por_grid);




    double tiempo = cpuSecond();

    switch(kernel){

        case 0:
            kernel_reduccion_intervalos<<<bloques_por_grid,hebras,sizeof(int)*bloques_por_grid>>>(device_vector, device_salida);
        break;

        case 1:
            kernel_reduccion_secuencial<<<bloques_por_grid,hebras,sizeof(int)*bloques_por_grid>>>(device_vector, device_salida);
        break;

        case 2:
            kernel_reduccion_desenrrollado_parcial<<<bloques_por_grid,hebras,sizeof(int)*bloques_por_grid>>>(device_vector, device_salida);
        break;

        case 3:
            kernel_reduccion_desenrrollado_total<<<bloques_por_grid,hebras,sizeof(int)*bloques_por_grid>>>(device_vector, device_salida);
        break;

    }

    tiempo = cpuSecond() - tiempo;

    hipMemcpy(resultado, device_salida, bloques_por_grid*sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    int max = maximo_local(resultado,bloques_por_grid);

    if(imprimir){
        cout << "Kernel: " << kernel << " Hebras: " << hebras << " Bloques: " << bloques_por_grid << "  Tiempo: " << tiempo << endl;
        cout << "El maximo obtenido al rellenar el ciclo es: " << maximo       << endl;
        cout << "El maximo obtenido al usar reduccion    es: " << max << endl;
    }else{
        cout << tiempo << " ";
    }

    /*
    Algoritmo secuencial
    double tiempo = cpuSecond();

    int max = maximo_local(vector,N);

    tiempo = cpuSecond() - tiempo;

    cout << tiempo << " ";
    */
    hipFree(device_salida);
    hipFree(device_vector);
    free(vector);
    free(resultado);

}
